/*
Zalozenia do sieci przeplywowej
	brak auto krawedzi
	miedzy kazda para wierzcholkow co najwyzej jedna skierowana krawedz
*/

#include <iostream>
#include <vector>
#include "flow.h"
using namespace std;

vector<vector<edge>> G;

int main() {
	ios_base::sync_with_stdio(false);
	hipSetDevice(3);
	int n,m,s,t;
	cin >> n >> m >> s >> t;
	G.resize(n+1);
	for (int i = 0; i < m; i++) {
		int a,b,c;
		cin >> a >> b >> c;
		add_edge(a,b,c,G);
	}
	compute_flow_gpu(n,m,s,t,G);
	compute_flow_cpu(n,m,s,t,G);
	compute_flow_gpu_on_cpu(n,m,s,t,G);
	
}
