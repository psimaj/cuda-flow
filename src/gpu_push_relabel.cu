#include "hip/hip_runtime.h"
#include <stdio.h>
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#include <iostream>
#include "flow.h"
using namespace std;

__device__ void relabel_gpu(node & u, int h) {
	u.height = h+1;
}

__device__ void push_gpu(node & u, node & v, edge & e, edge *E) {
	edge & back  = E[e.mate];
	int d = min(u.excess, e.capacity);
	atomicSub(&e.capacity,d);
	atomicAdd(&back.capacity,d);
	atomicSub(&u.excess,d);
	atomicAdd(&v.excess,d);
	// u.excess -= d;
	// v.excess += d;
	// e.capacity -= d;
	// back.capacity += d;

}

__global__ void push_relabel_kernel(int n, int s, int t, node *V, edge *E) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i == 0 || i > n || i == s || i == t) {
		return;
	}
	node &u = V[i];
	
	int CYCLE = 500;
	for (int z = 0; z < CYCLE; z++) {
		if (u.excess == 0 || u.height >= n) {
			continue;
		}
		int h = 2*n;
		int tmp = -1;
		for(int i = u.edges_begin; i < u.edges_end; i++) {
			edge & e = E[i];
			node & v = V[e.vertex];
			if (e.capacity == 0) {
				continue;
			}
			if (v.height < h) {
				h = v.height;
				tmp = i;
			}
		}
		edge & e = E[tmp];
		node & v = V[e.vertex];
		if (u.height == v.height + 1) {
			push_gpu(u,v,e,E);
		} else {
			relabel_gpu(u,v.height);
		}
	}
	
}

void global_relabeling_cpu(int s, int t, vector<node> &V, vector<edge> &E, int & excess_sum) {
	excess_sum = 0;
	int start = t;
	int start_h = 0;
	queue<int> q;
	V[start].height = start_h;
	q.push(start);
	while (!q.empty()) {
		int i = q.front();
		q.pop();
		node & u = V[i];

		if (i != s && i != t) {
			excess_sum += u.excess;
		}

		for (int i = u.edges_begin; i < u.edges_end; i++) {
			edge & e = E[i];
			edge & back = E[e.mate];
			node & v = V[e.vertex];
			if (back.capacity > 0 && v.height > u.height + 1) {
				v.height = u.height + 1;
				q.push(e.vertex);
			}
		}
	}
}

void cancel_bad_flow_cpu(int n, int s, int t, vector<node> &V, vector<edge> &E) {
	for (int i = 1; i <= n; i++) {
		node &u = V[i];
		for (int j = u.edges_begin; j < u.edges_end; j++) {
			edge &e = E[j];
			edge &back = E[e.mate];
			node &v = V[e.vertex];
			if (u.height > v.height + 1) {
				u.excess -= e.capacity;
				v.excess += e.capacity;
				back.capacity += e.capacity;
				e.capacity = 0;
			}
		}
	}
}

void compute_flow_gpu(int n, int s, int t, vector<node> &V, vector<edge> &E) {
	node *dev_V;
	edge *dev_E;
	gpuErrchk(hipMalloc(&dev_V, sizeof(node)*(n+1)));
	gpuErrchk(hipMalloc(&dev_E, sizeof(edge)*E.size()));

	gpuErrchk(hipMemcpy(dev_V, V.data(), sizeof(node)*(n+1), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_E, E.data(), sizeof(edge)*E.size(), hipMemcpyHostToDevice));


	int excess_sum = -1;
	while (excess_sum != 0) {
		gpuErrchk(hipMemcpy(dev_V, V.data(), sizeof(node)*(n+1), hipMemcpyHostToDevice));

		push_relabel_kernel<<<1,1024>>>(n,s,t,dev_V,dev_E);

		gpuErrchk(hipMemcpy(V.data(), dev_V, sizeof(node)*(n+1), hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(E.data(), dev_E, sizeof(edge)*E.size(), hipMemcpyDeviceToHost));

		for (int i = 1; i <= n; i++) {
			V[i].height = 2*n+1;
		}
		cancel_bad_flow_cpu(n,s,t,V,E);
		global_relabeling_cpu(s,t,V,E,excess_sum);
		//cout << excess_sum << endl;
	}
}	

void compute_flow_gpu(int n, int m, int s, int t, vector<vector<edge>> &G) {
	vector<node> V;
	vector<edge> E;
	init_graph(n,s,t,V,E,G);
	init_flow(n,s,t,V,E);
	compute_flow_gpu(n,s,t,V,E);
	cout << "Flow : " << V[t].excess << " gpu" << endl;
}
